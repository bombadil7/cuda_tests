#include "hip/hip_runtime.h"
#include <time.h>
#include <hip/hip_runtime.h>
#include "book.h"

void init_vector(int* v, int size)  {
    for (int i = 0; i < size; ++i)
        v[i] = i;
}

__global__ void add(int *a, int *b, int size) {
    int tid = blockIdx.x;
    if (tid < size)
        b[tid] += a[tid];
}

int main(void){
    time_t old_time, new_time;
    
    const int len =   15000000;
    //const int len = 500000000;

    int* a = (int*) malloc(len * sizeof(int));
    int* b = (int*) malloc(len * sizeof(int));
    int* c = (int*) malloc(len * sizeof(int));

    int *d_a, *d_b;
    HANDLE_ERROR( hipMalloc((void**) &d_a, len * sizeof(int)) );
    HANDLE_ERROR( hipMalloc((void**) &d_b, len * sizeof(int)) );
 //   HANDLE_ERROR( hipMalloc((void**) &d_c, len * sizeof(int)) );


    //init_vector(a, len);
    //init_vector(b, len);
    for (int i = 0; i < len; ++i){
        a[i] = -i;
        b[i] = i * i;
    }

    HANDLE_ERROR(
        hipMemcpy(d_a, a, len * sizeof(int), 
            hipMemcpyHostToDevice));
    HANDLE_ERROR(
        hipMemcpy(d_b, b, len * sizeof(int),
            hipMemcpyHostToDevice));

    time(&old_time);
    add<<<65000, 1>>>(d_a, d_b, len);
    time(&new_time);

    HANDLE_ERROR(
        hipMemcpy(c, d_b, len * sizeof(int),
            hipMemcpyDeviceToHost));

    printf("Resulting array size is %d, addition took %ld seconds \n", len, new_time - old_time);

    free(a);
    free(b);
    free(c);

    hipFree(d_a);
    hipFree(d_b);
  //  hipFree(d_c);
    return 0;
}