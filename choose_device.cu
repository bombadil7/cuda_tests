#include <iostream>
#include "book.h"

using namespace std;


int main(void) {
    hipDeviceProp_t prop;
    int dev;

    HANDLE_ERROR( hipGetDevice(&dev) );

    printf("ID of current CUDA device: %d\n", dev);

    memset( &prop, 0, sizeof( hipDeviceProp_t ) );
    prop.major = 3;
    prop.minor = 0;

    HANDLE_ERROR( hipChooseDevice( &dev, &prop ) );

    printf("ID of CUDA device closest to revision 3.0: %d\n", dev);

    HANDLE_ERROR( hipSetDevice( dev ) );


    return 0;
}

