#include "hip/hip_runtime.h"
#include "book.h"
#include <time.h>

#define imin(a, b) (a < b ? a : b)

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N+threadsPerBlock-1) / threadsPerBlock);

__global__ void dot(float *a, float *b, float *c) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while (tid < N){
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;
    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0){
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}


int main(void) {
    struct timespec old_time, new_time;
    unsigned long int oldNs, newNs; 

    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;

    a = (float*) malloc(N * sizeof(float));
    b = (float*) malloc(N * sizeof(float));
    partial_c = (float*) malloc(blocksPerGrid * sizeof(float));

    HANDLE_ERROR( hipMalloc((void**) &dev_a, N * sizeof(float)) );
    HANDLE_ERROR( hipMalloc((void**) &dev_b, N * sizeof(float)) );
    HANDLE_ERROR( hipMalloc((void**) &dev_partial_c, blocksPerGrid * sizeof(float)) );

    for(int i = 0; i < N; ++i){
        a[i] = i;
        b[i] = i * 2;
    }

    HANDLE_ERROR( hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice) );

    clock_gettime(CLOCK_MONOTONIC, &old_time);
    dot <<<blocksPerGrid, threadsPerBlock>>> (dev_a, dev_b, dev_partial_c);
    hipDeviceSynchronize();
    HANDLE_ERROR( hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost) );

    clock_gettime(CLOCK_MONOTONIC, &new_time);
    oldNs = old_time.tv_sec * 1000000000ull + old_time.tv_nsec;
    newNs = new_time.tv_sec * 1000000000ull + new_time.tv_nsec;
    float dt = (newNs - oldNs) * 0.000000001f;
    printf("Original vector sizes were %d, dot product took %0.6f seconds \n", N, dt);

    //HANDLE_ERROR( hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost) );

    c = 0;
    for (int i = 0; i < blocksPerGrid; ++i) {
        c += partial_c[i];
    }

    #define sum_squares(x) (x * (x + 1) * (2 * x + 1) / 6)
    printf("Does GPU value %.6g = %.6g?\n", c, 2 * sum_squares( (float) (N - 1) ));

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_partial_c);

    free(a);
    free(b);
    free(partial_c);

    return 0;
}